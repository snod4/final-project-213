#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <openssl/md5.h>
#include <hip/hip_runtime.h>
#include "md5.cu"

#define NUM_THREADS 128
#define MAX_USERNAME_LENGTH 64

#define PASSWORD_LENGTH 6
#define SIXTH_POWER  (26 * 26 * 26 * 26 * 26* 26)
#define FIFTH_POWER (26 * 26 * 26 * 26 * 26)
#define FOURTH_POWER (26 * 26 * 26 * 26)
#define THIRD_POWER (26 * 26 * 26)
#define SECOND_POWER (26 * 26)
#define FIRST_POWER 26

#define HASH_LENGTH 32

__device__ size_t POWER_ARR[] = {1, FIRST_POWER, SECOND_POWER, THIRD_POWER, FOURTH_POWER, FIFTH_POWER};


typedef struct hashInfo{
  char  password[7];
  uint  hash[4];
  int found = 0;
}hashInfo_t;


//Crack is a function that runs on the GPU and brute forces given hashes.
__global__  void crack(hashInfo_t * hashData, int length){
  //get string permuation
  size_t tempNum =((size_t) blockIdx.x) * ((size_t) NUM_THREADS) +((size_t) threadIdx.x);
  char word[] = "aaaaaa";

  //Generate the permutation for the given thread/core
  for(int i = PASSWORD_LENGTH - 1; i >= 0; i--){
    size_t temp =  tempNum/(POWER_ARR[i]);
    word[5 - i] += temp;
    tempNum = tempNum % POWER_ARR[i];
  }   

  //Calculate the hash with the function md5.
  uint candidate_hash[4];
  md5((uint*)word, candidate_hash);

  //Compare the provided hash to the calculated candidate hash.
  for(int j = 0; j < length; j++){
    if(!(hashData[j].found)
       && candidate_hash[0] == hashData[j].hash[0]
       && candidate_hash[1] == hashData[j].hash[1]
       && candidate_hash[2] == hashData[j].hash[2]
       && candidate_hash[3] == hashData[j].hash[3]){
      memcpy(hashData[j].password, word, PASSWORD_LENGTH+1);
      hashData[j].found = 1;
      break;
    }
  }
  
}

//add hash to hash table
/* void addToTable(hashInfo * table, char * hash){ */
/*   hashInfo_t * temp = (hashInfo_t *) malloc(sizeof(hashInfo_t)); */
/*   strncpy(temp->hash, hash, PASSWORD_LENGTH); */
/*   temp->next = NULL; */
/*   temp->password = NULL; */
  
/*   if(table[hash[0] - 48 ]== NULL){ */
/*     table[hash[0] - 48] = temp; */
/*   } */
/*   else{ */
/*     temp->next =  table[hash[0] - 48]; */
/*     table[hash[0] - 48] = temp; */
/*   } */
/* } */

int main(int argv, char* args[]){
  /*  hashInfo_t * hashTable[74];
      int count = 0;
      //get hashes in here -- add them -- count them //
  
      hashInfo_t * gpu_hashTable;
 
      //ISSUE IN COPYING A LINKED LIST TO THE GPU
      */
  FILE * file = fopen("outputFile.txt", "r");/////////////CHANGE THIS TO ARGV EVENTUALLY/////////////
  int NUM_INPUT = 0;
  fscanf(file, "%d", &NUM_INPUT);
  printf("Number of Input: %d\n", NUM_INPUT);
  
  int number_of_blocks = (308915776+NUM_THREADS)/NUM_THREADS;
  hashInfo_t arr[NUM_INPUT];
  
  
  uint hash[4];
  int count = 0;
  char trash_can[7];

  //Grab the input hashes from a file specified by the user in argv[1].
  while(fscanf(file, "%s", trash_can) != EOF){
    for(int i = 0; i < 4; i++){
      fscanf(file, "%u", &hash[i]);
    }
    memcpy(arr[count].hash, hash, sizeof(uint)*4);
    count++;
  }

  //Create the data structure to pass to the GPU
  hashInfo_t * gpu_arr;
  if(hipMalloc(&gpu_arr, sizeof(hashInfo_t) * NUM_INPUT) != hipSuccess){
    perror("Cuda Malloc Failed\n");
    exit(2);
  }

  //Copy over our provided hashes in arr to the GPU_arr for analysis.
  if(hipMemcpy(gpu_arr, arr, sizeof(hashInfo_t) * NUM_INPUT, hipMemcpyHostToDevice) != hipSuccess){
    perror("Cuda CPU to GPU memcpy Failed\n");
    exit(2);
  }

  //Crack the provided hashes on the GPU
  crack<<<number_of_blocks, NUM_THREADS>>>(gpu_arr,NUM_INPUT);

  //Ensure all CUDA threads have terminated
  if(hipDeviceSynchronize() != hipSuccess){
    perror("CUDA Thread Synchronization Error\n");
    exit(2);
  }

  //Copy back the cracked passwords from the GPU.
  if(hipMemcpy(arr, gpu_arr, sizeof(hashInfo_t) * NUM_INPUT, hipMemcpyDeviceToHost) != hipSuccess){
    perror("Cuda GPU to CPU memcpy Failed\n");
    exit(2);
  }

  //Print the cracked passwords. Eventually we should delete this and automate
  //password cracking sucess when we scale up the amount of passwords to crack
  for(int i = 0; i < NUM_INPUT; i++){
    printf("Password: %s, Hash:", arr[i].password);
    for(int g = 0; g < 4; g++){
      printf("%u ", arr[i].hash[g]);
    }
    printf("\n");
  }

  
  

  
  
  return 0;
}
