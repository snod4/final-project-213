#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <openssl/md5.h>
#include <hip/hip_runtime.h>
#include "md5.cu"
#include <unistd.h>

#define NUM_THREADS 128
#define MAX_USERNAME_LENGTH 64
#define DEPTH 100
#define PASSWORD_LENGTH 6
#define SIXTH_POWER  (26 * 26 * 26 * 26 * 26* 26)
#define FIFTH_POWER (26 * 26 * 26 * 26 * 26)
#define FOURTH_POWER (26 * 26 * 26 * 26)
#define THIRD_POWER (26 * 26 * 26)
#define SECOND_POWER (26 * 26)
#define FIRST_POWER 26
#define NUMBER_OF_BINS 256

//these denote whether or not the hash table is at capacity in a certain bin in a given addToHashTable call
#define SUCCESS 43
#define FAILURE 21

#define HASH_LENGTH 32 

/////////////////////////////// GLOBALS AND DATA STRUCTURES  /////////////////////////////////

__device__ size_t POWER_ARR[] = {1, FIRST_POWER, SECOND_POWER, THIRD_POWER, FOURTH_POWER, FIFTH_POWER};

__device__ int num_cracked = 0;


typedef struct hashInfo{
  char  password[7]; 
  uint  hash[4];
  int empty = 1; //denotes whether a hashInfo_t has a hash value or not
  int length; //gives the number of items in a bin of a hash table
}hashInfo_t;

/////////////////////////////// GPU FUNCTIONS  /////////////////////////////////

/*
  Determines if the hash dentoed by hash exists in the hash table denoted by table. 
  If it does, the function returns 1, otherwise it returns 0
  Password signifes the word to be inserted into the hash table if it matches
*/
__device__ int isHash(hashInfo_t * table, uint * hash, char * password){
  //get the bin of the hash by taking the first byte of the first uint of the hash
  unsigned char byte;
  byte = (hash[0]&0xFF);
  
  //loops through the bin -- first element of the bin is empty but denotes the size of the bin
  for(int i = 1; i < table[DEPTH*byte].length+1; i++){ 
    int index = DEPTH*byte+i;
    if(!table[index].empty
       && table[index].hash[0] == hash[0]
       && table[index].hash[1] == hash[1]
       && table[index].hash[2] == hash[2]
       && table[index].hash[3] == hash[3]){
      num_cracked++;
      //copies password to the password field of the hashInfo_t with the matching hash
      memcpy(table[index].password, password, PASSWORD_LENGTH*sizeof(char)); 
      return 1;
       
    }
     
  }
  return 0;
}


/*
  The kernel, crack, runs on the gpu and brute force cracks 6 character, alphabetic passwords
  hashData denotes a hash table
*/
__global__  void crack(hashInfo_t * hashData){
  
  //get string permuation
  size_t tempNum =((size_t) blockIdx.x) * ((size_t) NUM_THREADS) +((size_t) threadIdx.x); 

  //starts at the lowest value
  char word[] = "aaaaaa"; 

  //Generate the permutation for the given thread/core
  for(int i = PASSWORD_LENGTH - 1; i >= 0; i--){
    size_t temp =  tempNum/(POWER_ARR[i]);
    word[5 - i] += temp;
    tempNum = tempNum % POWER_ARR[i];
  }   

  //Calculate the hash with the function md5.
  uint candidate_hash[4];
  md5((uint*)word, candidate_hash);

  //Checks for matching hash and inserts word into hashData if candidate_hash matches
  isHash(hashData, candidate_hash, word);
  return;
    
}


///////////////////////// HASH TABLE FUNCTIONS ////////////////////////////////

//add hash to hash table
//uses first byte of hash to index hash in hashTable
//returns SUCCESS if there is space in the bin denoted by the first byte for the hash
//returns FAILURE otherwise
int addToTable(hashInfo_t * table, uint * hash){
  //get the bin of the hash by taking the first byte of the first uint of the hash
  unsigned char byte;
  byte = (hash[0]&0xFF);


  //handles insertion and length incrementing in the bin given by byte
  if(table[DEPTH*byte].length == 0){
    table[DEPTH*byte+1].hash[0] = hash[0];
    table[DEPTH*byte+1].hash[1] = hash[1];
    table[DEPTH*byte+1].hash[2] = hash[2];
    table[DEPTH*byte+1].hash[3] = hash[3];
         
    table[DEPTH*byte+1].empty = 0;
    table[DEPTH*byte].length++;
    return SUCCESS;
  }
  
  int placement = table[DEPTH*byte].length+1;
  if(placement - 1 < DEPTH  - 1){
    table[DEPTH*byte+placement].hash[0] = hash[0];
    table[DEPTH*byte+placement].hash[1] = hash[1];
    table[DEPTH*byte+placement].hash[2] = hash[2];
    table[DEPTH*byte+placement].hash[3] = hash[3];
    table[DEPTH*byte].length++;
    table[DEPTH*byte+placement].empty = 0;
    return SUCCESS;
  }

  return FAILURE;
}

/*
  Prints the contents of the hash table in the order they 
  are stored in the hash table, which is just an array.
  It does not preserve the order of the hashes in the outputFile.txt.
*/
void printHashTable(hashInfo_t  * table){
  //This nested loop takes into account that all bins the hash table are not filled
  //and, as such, avoids the empty bins
  for(int r = 0; r < NUMBER_OF_BINS; r++){
    for( int c = 1; c < table[r*DEPTH].length + 1; c++){
      printf("Password: %s, Hash: ",table[DEPTH*r+c].password);
      for(int g = 0; g < 4; g++){
        printf("%u ", table[DEPTH*r+c].hash[g]);
      }
      printf("\n");
    }
  }
}


/*
  Initializes the first element of each bin, which holds the number of items in each bin given by length
*/
void initializeTable(hashInfo_t * table){
  for(int i = 0; i < NUMBER_OF_BINS*DEPTH; i++){
    table[i].empty = 1; //true
    table[i].length = 0; //nothing in the bin
  }
  
}


/////////////////////////// MAIN ////////////////////////////

int main(int argv, char* args[]){
 
  //value returned by fscanf
  int eof = 0;

  int counter = 0;
  
  //opens file generated from make-hash-file
  FILE * file = fopen("outputFile.txt", "r"); 
  printf("OPENED FILE\n");
  
  //calculates number of blocks needed assuming every thread computes one
  //possible six-character, alpabetic string permuation.
  int number_of_blocks = (SIXTH_POWER+NUM_THREADS)/NUM_THREADS;
  
  
  //gets hash from file
  uint hash[4];
  //gets password which is also stored in file but is not needed for this code -- ignores it
  char trash_can[7];

  //Grab the input hashes from a file ideally specified by the user but, for our purposes, specified by outputFile.txt instead
  //If one of the bins of the hash table fills ups completely, the items in the table must be processed. This condition ensures
  //that even if it takes multiple passes, all items in file will be processed
  while(eof != EOF){
    
    //allocates space for the hash table
    hashInfo_t *  hashTable = (hashInfo_t *)malloc(sizeof(hashInfo_t)*NUMBER_OF_BINS * DEPTH);
    initializeTable(hashTable);
   
    //Processes previous item if previous addToTable call returned FAILURE
    if(counter != 0){
      addToTable(hashTable,hash);
      counter++;
    }

    //Reads in input, ignoring passwords and storing hashes
    while((eof = fscanf(file, "%s", trash_can)) != EOF){
      for(int i = 0; i < 4; i++){
        fscanf(file, "%u", &hash[i]);
      }

      //This condition triggers when one of the bins in the hash table is full
      if(addToTable(hashTable, hash) == FAILURE){
        break;
      }
      counter++;
    }

    //Creates the data structure to pass to the GPU
    hashInfo_t  * gpu_hashTable;
    if(hipMalloc(&gpu_hashTable, sizeof(hashInfo_t) * NUMBER_OF_BINS * DEPTH) != hipSuccess){
      perror("Cuda Malloc Failed\n");
      exit(2);
    }

    //Copy over our provided hashes in hashTable to the gpu_hashTable for analysis.
    if(hipMemcpy(gpu_hashTable, hashTable, sizeof(hashInfo_t) * NUMBER_OF_BINS * DEPTH, hipMemcpyHostToDevice) != hipSuccess){
      perror("Cuda CPU to GPU memcpy Failed\n");
      exit(2);
    }

    //Crack the provided hashes on the GPU
    printf("Begin Cracking of this Portion\n");
    
    crack<<<number_of_blocks, NUM_THREADS>>>(gpu_hashTable);
     

    //Ensure all CUDA threads have terminated
    if(hipDeviceSynchronize() != hipSuccess){
      perror("CUDA Thread Synchronization Error\n");
      exit(2);
    }
    printf("End Cracking of this Portion\n");

    //Copy back the cracked passwords from the GPU.
    if(hipMemcpy(hashTable, gpu_hashTable, sizeof(hashInfo_t) * NUMBER_OF_BINS * DEPTH, hipMemcpyDeviceToHost) != hipSuccess){
      perror("Cuda GPU to CPU memcpy Failed\n");
      exit(2);
    }

    //Prints the cracked passwords. Could easily be written to a file instead
    printHashTable(hashTable);

    //frees hashTable for this portion of the cracking process
    free(hashTable);
  }
  
  

  
  
  return 0;
}
